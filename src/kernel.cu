#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "Config.h"
#include "FileMgr.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>

#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <math.h>

struct normal_deviate {

    float mu, sigma;

    __host__ __device__ normal_deviate(float _mu = 0.0f, float _sigma = 1.0f) : mu(_mu), sigma(_sigma) {};

    __device__ float operator()(unsigned int n) {

        thrust::default_random_engine engine;
        thrust::normal_distribution<float> dist(mu, sigma);
        engine.discard(n);

        return dist(engine);

    }

};

int main(int argc, char** argv)
{

    // Get input parameters
    if (argc != 2) {
        std::cerr << "TinyDisp - A simple, local airflow visualizer" << std::endl;
        std::cerr << std::endl;
        std::cerr << "Usage:" << std::endl;
        std::cerr << std::endl;
        std::cerr << "    td <RunConfiguration>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "Copyright 2020 by Servizi Territorio srl" << std::endl;
        std::cerr << "                  This is open-source software, covered by the MIT license" << std::endl;
        return 1;
    }
    std::string sCfgFile = argv[1];

    // Gather configuration (and meteo data)
    Config tCfg(sCfgFile);

    // Particle pool
    int iNumPart  = tCfg.GetParticlePoolSize();
    int iNextPart = 0;  // For indexing the generation circular buffer
    thrust::device_vector<int> ivPartTimeStamp(iNumPart); // Time stamp at emission time - for reporting - host-only
    thrust::device_vector<float> rvPartX(iNumPart);
    thrust::device_vector<float> rvPartY(iNumPart);
    thrust::device_vector<float> rvPartU(iNumPart);
    thrust::device_vector<float> rvPartV(iNumPart);
    thrust::device_vector<float> rvN1(iNumPart);
    thrust::device_vector<float> rvN2(iNumPart);
    thrust::device_vector<float> rvX1(iNumPart);
    thrust::device_vector<float> rvX2(iNumPart);
    thrust::device_vector<float> rvDeltaU(iNumPart);
    thrust::device_vector<float> rvDeltaV(iNumPart);
    thrust::host_vector<float>   rvCellX(iNumPart);
    thrust::host_vector<float>   rvCellY(iNumPart);
    thrust::host_vector<float>   rvTempX(iNumPart);
    thrust::host_vector<float>   rvTempY(iNumPart);

    // Initialize the particles' time stamp to -1, to mean "not yet assigned" the parallel vay
    thrust::fill(ivPartTimeStamp.begin(), ivPartTimeStamp.end(), -1);

    // Main loop: iterate over meteo data
    std::string sOutFileName = tCfg.GetOutputFile();
    auto fOut = std::fstream(sOutFileName, std::ios::out | std::ios::binary);
    fOut.write((char*)&iNumPart, sizeof(int));
    int iNumData = tCfg.GetNumMeteoData();
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    unsigned int iIteration = 0;
    int iFirstTimeStamp = 0;
    int iNextRandomBlock = 0;
    for (auto i = 0; i < iNumData; i++) {

        // Get current meteorology
        int iTimeStamp;
        float rU;
        float rV;
        float rStdDevU;
        float rStdDevV;
        float rCovUV;
        bool lOk = tCfg.GetMeteo(i, iTimeStamp, rU, rV, rStdDevU, rStdDevV, rCovUV);
        if (iIteration == 0) {
            iFirstTimeStamp = iTimeStamp;
        }

        // Emit new particles
        thrust::fill(ivPartTimeStamp.begin() + iNextPart, ivPartTimeStamp.begin() + iNextPart + tCfg.GetNumNewParticles(), iTimeStamp);
        thrust::fill(rvPartX.begin() + iNextPart, rvPartX.begin() + iNextPart + tCfg.GetNumNewParticles(), 0.0f);
        thrust::fill(rvPartY.begin() + iNextPart, rvPartY.begin() + iNextPart + tCfg.GetNumNewParticles(), 0.0f);
        thrust::fill(rvPartU.begin() + iNextPart, rvPartU.begin() + iNextPart + tCfg.GetNumNewParticles(), rU);
        thrust::fill(rvPartV.begin() + iNextPart, rvPartV.begin() + iNextPart + tCfg.GetNumNewParticles(), rV);
        iNextPart += tCfg.GetNumNewParticles();
        if (iNextPart >= iNumPart) {
            iNextPart = 0;
        }

        // Generate bivariate normal deviates
        // -1- First of all, generate two sets of random normals, with mu=0 and sigma=1
        thrust::transform(
            index_sequence_begin + iNextRandomBlock,
            index_sequence_begin + iNextRandomBlock + iNumPart,
            rvN1.begin(),
            normal_deviate(0.0f, 1.0f)
        );
        iNextRandomBlock += iNumPart;
        thrust::transform(
            index_sequence_begin + iNextRandomBlock,
            index_sequence_begin + iNextRandomBlock + iNumPart,
            rvN2.begin(),
            normal_deviate(0.0f, 1.0f)
        );
        iNextRandomBlock += iNumPart;
        iIteration++;
        // -1- Transform the two independent samples in a 2D bivariate sample
        float rho;
        float lambda;
        if (rStdDevU > 0.f && rStdDevV > 0.f) {
            rho = rCovUV / (rStdDevU * rStdDevV);
            rho = rho < -1.f ? -1.f : rho;
            rho = rho >  1.f ?  1.f : rho;
            lambda = (rStdDevV / rStdDevU) * rho;
        }
        else {
            rho = 0.f;
            lambda = 0.f;
        }
        float nu = sqrtf((1.0f - rho * rho) * rStdDevV * rStdDevV);
        // x1 = v1[i] = mu1 + sigma1 * standardnormal(generator)
        rvX1 = rvN1;
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rStdDevU), rvX1.begin(), thrust::multiplies<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rU), rvX1.begin(), thrust::plus<float>());
        rvPartU = rvX1;
        // v2[i]=mu2+lambda*(x1-mu1)+nu*standardnormal(generator)
        rvX2 = rvN2;
        thrust::transform(rvX2.begin(), rvX2.end(), thrust::make_constant_iterator(nu), rvX2.begin(), thrust::multiplies<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rU), rvX1.begin(), thrust::minus<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(lambda), rvX1.begin(), thrust::multiplies<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), rvX2.begin(), rvX1.begin(), thrust::plus<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rV), rvX1.begin(), thrust::plus<float>());
        rvPartV = rvX1;

        // Move particles
        float rDeltaT = tCfg.GetTimeStep();
        thrust::transform(rvPartU.begin(), rvPartU.end(), rvDeltaU.begin(), rvPartU.begin(), thrust::plus<float>());
        rvX1 = rvPartU;
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rDeltaT), rvX1.begin(), thrust::multiplies<float>());
        thrust::transform(rvPartX.begin(), rvPartX.end(), rvX1.begin(), rvPartX.begin(), thrust::plus<float>());
        thrust::transform(rvPartV.begin(), rvPartV.end(), rvDeltaV.begin(), rvPartV.begin(), thrust::plus<float>());
        rvX2 = rvPartV;
        thrust::transform(rvX2.begin(), rvX2.end(), thrust::make_constant_iterator(rDeltaT), rvX2.begin(), thrust::multiplies<float>());
        thrust::transform(rvPartY.begin(), rvPartY.end(), rvX2.begin(), rvPartY.begin(), thrust::plus<float>());

        // Append particles to pool
        rvTempX = rvPartX;
        rvTempY = rvPartY;
        int iNumActivePart = 0;
        for (auto i = 0; i < iNumPart; ++i) {
            if (ivPartTimeStamp[i] >= 0) {
                if (tCfg->GetMinX() <= rvTempX[i] && rvTempX[i] <= -tCfg->GetMaxX() && tCfg->GetMinY() <= rvTempY[i] && rvTempY[i] <= -tCfg->GetMaxY()) {
                    ++iNumActivePart;
                }
            }
        }
        fOut.write((char*)&iNumActivePart, sizeof(int));
        for (auto i = 0; i < iNumPart; ++i) {
            if (ivPartTimeStamp[i] >= 0) {
                if (tCfg->GetMinX() <= rvTempX[i] && rvTempX[i] <= -tCfg->GetMaxX() && tCfg->GetMinY() <= rvTempY[i] && rvTempY[i] <= -tCfg->GetMaxY()) {
                    fOut.write((char*)&rvTempX[i], sizeof(float));
                    fOut.write((char*)&rvTempY[i], sizeof(float));
                    fOut.write((char*)&ivPartTimeStamp[i], sizeof(int));
                }
            }
        }

        // Inform users of the progress
        std::cout << iIteration << " of " << iNumData << ", " << rU << ", " << rV << ", " << rStdDevU << ", " << rStdDevV << ", " << rCovUV << std::endl;

    }

    // Release OS resources
    fOut.close();

    // Deallocate manually thrust resources
    // -1- Release count matrices
    delete imNumPartsInCell;
    // -1- Reclaim workspace
    ivPartTimeStamp.clear();
    rvPartX.clear();
    rvPartY.clear();
    rvPartU.clear();
    rvPartV.clear();
    rvN1.clear();
    rvN2.clear();
    rvX1.clear();
    rvX2.clear();
    rvDeltaU.clear();
    rvDeltaV.clear();
    rvCellX.clear();
    rvCellY.clear();
    rvTempX.clear();
    rvTempY.clear();
    // -1- Clear any other resources
    ivPartTimeStamp.shrink_to_fit();
    rvPartX.shrink_to_fit();
    rvPartY.shrink_to_fit();
    rvPartU.shrink_to_fit();
    rvPartV.shrink_to_fit();
    rvN1.shrink_to_fit();
    rvN2.shrink_to_fit();
    rvX1.shrink_to_fit();
    rvX2.shrink_to_fit();
    rvDeltaU.shrink_to_fit();
    rvDeltaV.shrink_to_fit();
    rvCellX.shrink_to_fit();
    rvCellY.shrink_to_fit();
    rvTempX.shrink_to_fit();
    rvTempY.shrink_to_fit();

    // Leave
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        std::cout << "hipDeviceReset failed!" << std::endl;
        return 1;
    }

    std::cout << "*** End Job ***" << std::endl;

    return 0;
}
