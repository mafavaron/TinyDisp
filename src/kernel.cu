#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "Config.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/functional.h>
#include <stdio.h>

#include <iostream>
#include <math.h>

struct normal_deviate {

    float mu, sigma;

    __host__ __device__ normal_deviate(float _mu = 0.0f, float _sigma = 1.0f) : mu(_mu), sigma(_sigma) {};

    __device__ float operator()(unsigned int n) {

        thrust::default_random_engine engine;
        thrust::normal_distribution<float> dist(mu, sigma);
        engine.discard(n);

        return dist(engine);

    }

};

int main(int argc, char** argv)
{

    // Get input parameters
    if (argc != 2) {
        std::cerr << "TinyDisp - A simple, local airflow visualizer" << std::endl;
        std::cerr << std::endl;
        std::cerr << "Usage:" << std::endl;
        std::cerr << std::endl;
        std::cerr << "    td <RunConfiguration>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "Copyright 2020 by Servizi Territorio srl" << std::endl;
        std::cerr << "                  This is open-source software, covered by the MIT license" << std::endl;
        return 1;
    }
    std::string sCfgFile = argv[1];

    // Gather configuration (and meteo data)
    Config tCfg(sCfgFile);

    // Particle pool
    int iNumPart  = tCfg.GetParticlePoolSize();
    int iNextPart = 0;  // For indexing the generation circular buffer
    thrust::device_vector<int> ivPartTimeStamp(iNumPart); // Time stamp at emission time - for reporting - host-only
    thrust::device_vector<float> rvPartX(iNumPart);
    thrust::device_vector<float> rvPartY(iNumPart);
    thrust::device_vector<float> rvPartU(iNumPart);
    thrust::device_vector<float> rvPartV(iNumPart);
    thrust::device_vector<float> rvN1(iNumPart);
    thrust::device_vector<float> rvN2(iNumPart);
    thrust::device_vector<float> rvX1(iNumPart);
    thrust::device_vector<float> rvX2(iNumPart);
    thrust::device_vector<float> rvDeltaU(iNumPart);
    thrust::device_vector<float> rvDeltaV(iNumPart);
    thrust::host_vector<float>   rvCellX(iNumPart);
    thrust::host_vector<float>   rvCellY(iNumPart);

    // Main loop: iterate over meteo data
    std::string sOutFileName = tCfg.GetOutputFile();
    FILE* fOut = fopen(sOutFileName.c_str(), "wb");
    int n = tCfg.GetCellsPerEdge();
    auto imNumPartsInCell = new unsigned int[n * n];
    auto rmConc = new float[n * n];
    int iNumData = tCfg.GetNumMeteoData();
    thrust::counting_iterator<unsigned int> index_sequence_begin(0);
    unsigned int iIteration = 0;
    for (auto i = 0; i < iNumData; i++) {

        // Get current meteorology
        int iTimeStamp;
        float rU;
        float rV;
        float rStdDevU;
        float rStdDevV;
        float rCovUV;
        bool lOk = tCfg.GetMeteo(i, iTimeStamp, rU, rV, rStdDevU, rStdDevV, rCovUV);

        // Emit new particles
        thrust::fill(ivPartTimeStamp.begin() + iNextPart, ivPartTimeStamp.begin() + iNextPart + tCfg.GetNumNewParticles(), iTimeStamp);
        thrust::fill(rvPartX.begin() + iNextPart, rvPartX.begin() + iNextPart + tCfg.GetNumNewParticles(), 0.0f);
        thrust::fill(rvPartY.begin() + iNextPart, rvPartY.begin() + iNextPart + tCfg.GetNumNewParticles(), 0.0f);
        thrust::fill(rvPartU.begin() + iNextPart, rvPartU.begin() + iNextPart + tCfg.GetNumNewParticles(), rU);
        thrust::fill(rvPartV.begin() + iNextPart, rvPartV.begin() + iNextPart + tCfg.GetNumNewParticles(), rV);
        iNextPart += tCfg.GetNumNewParticles();
        if (iNextPart >= iNumPart) {
            iNextPart = 0;
        }

        // Generate bivariate normal deviates
        // -1- First of all, generate two sets of random normals, with mu=0 and sigma=1
        thrust::transform(
            index_sequence_begin + iIteration * iNumPart,
            index_sequence_begin + iIteration * (iNumPart + 1),
            rvN1.begin(),
            normal_deviate(0.0f, 1.0f)
        );
        thrust::transform(
            index_sequence_begin + iIteration * (iNumPart + 2),
            index_sequence_begin + iIteration * (iNumPart + 3),
            rvN2.begin(),
            normal_deviate(0.0f, 1.0f)
        );
        iIteration++;
        // -1- Transform the two independent samples in a 2D bivariate sample
        float rho;
        if (rStdDevU > 0.f && rStdDevV > 0.f) {
            rho = rCovUV / (rStdDevU * rStdDevV);
        }
        else {
            rho = 0.f;
        }
        float lambda = (rStdDevV / rStdDevU) * rho;
        float nu = sqrtf((1.0f - rho * rho) * rStdDevV * rStdDevV);
        rvX1 = rvN1;
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rStdDevU), rvX1.begin(), thrust::multiplies<float>());
        rvDeltaU = rvX1;
        rvX2 = rvN2;
        thrust::transform(rvX2.begin(), rvX2.end(), thrust::make_constant_iterator(nu), rvX2.begin(), thrust::multiplies<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rU), rvX1.begin(), thrust::minus<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(lambda), rvX1.begin(), thrust::multiplies<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), rvX2.begin(), rvX1.begin(), thrust::plus<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rV), rvX1.begin(), thrust::plus<float>());
        rvDeltaV = rvX1;

        // Move particles
        float rDeltaT = tCfg.GetTimeStep();
        thrust::transform(rvPartU.begin(), rvPartU.end(), rvDeltaU.begin(), rvPartU.begin(), thrust::plus<float>());
        rvX1 = rvPartU;
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(rDeltaT), rvX1.begin(), thrust::multiplies<float>());
        thrust::transform(rvPartX.begin(), rvPartX.end(), rvX1.begin(), rvPartX.begin(), thrust::plus<float>());
        thrust::transform(rvPartV.begin(), rvPartV.end(), rvDeltaV.begin(), rvPartV.begin(), thrust::plus<float>());
        rvX2 = rvPartV;
        thrust::transform(rvX2.begin(), rvX2.end(), thrust::make_constant_iterator(rDeltaT), rvX2.begin(), thrust::multiplies<float>());
        thrust::transform(rvPartY.begin(), rvPartY.end(), rvX2.begin(), rvPartY.begin(), thrust::plus<float>());

        // Count cell contents
        rvX1 = rvPartX;
        rvX2 = rvPartY;
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(tCfg.GetMinX()), rvX1.begin(), thrust::minus<float>());
        thrust::transform(rvX1.begin(), rvX1.end(), thrust::make_constant_iterator(tCfg.GetCellSize()), rvX1.begin(), thrust::divides<float>());
        thrust::transform(rvX2.begin(), rvX2.end(), thrust::make_constant_iterator(tCfg.GetMinY()), rvX2.begin(), thrust::minus<float>());
        thrust::transform(rvX2.begin(), rvX2.end(), thrust::make_constant_iterator(tCfg.GetCellSize()), rvX2.begin(), thrust::divides<float>());
        rvCellX = rvX1;
        rvCellY = rvX2;
        for (int iy = 0; iy < n; iy++) {
            for (int ix = 0; iy < n; iy++) {
                imNumPartsInCell[n * iy + ix] = 0U;
            }
        }
        for (int j = 0; j < rvCellX.size(); j++) {
            int ix = (int)rvCellX[j];
            int iy = (int)rvCellY[j];
            if (0 <= ix && ix < n && 0 <= iy && iy < n) {
                ++imNumPartsInCell[n * iy + ix];
            }
        }
        int iTotParticles = 0;
        for (int j = 0; j < n * n; j++) {
            iTotParticles += imNumPartsInCell[j];
        }
        float rTotParticles = iTotParticles;
        for (int j = 0; j < n * n; j++) {
            rmConc[j] = (float)imNumPartsInCell[j] / rTotParticles;
        }
        fwrite((void*)rmConc, sizeof(float), n * n, fOut);

        // Inform users of the progress
        std::cout << iIteration << ", " << rU << ", " << rV << ", " << rStdDevU << ", " << rStdDevV << ", " << rCovUV << std::endl;
    }

    // Release OS resources
    fclose(fOut);

    // Deallocate manually thrust resources
    // -1- Release count matrices
    delete rmConc;
    delete imNumPartsInCell;
    // -1- Reclaim workspace
    ivPartTimeStamp.clear();
    rvPartX.clear();
    rvPartY.clear();
    rvPartU.clear();
    rvPartV.clear();
    rvN1.clear();
    rvN2.clear();
    rvX1.clear();
    rvX2.clear();
    rvCellX.clear();
    rvCellY.clear();
    // -1- Clear any other resources
    ivPartTimeStamp.shrink_to_fit();
    rvPartX.shrink_to_fit();
    rvPartY.shrink_to_fit();
    rvPartU.shrink_to_fit();
    rvPartV.shrink_to_fit();
    rvN1.shrink_to_fit();
    rvN2.shrink_to_fit();
    rvX1.shrink_to_fit();
    rvX2.shrink_to_fit();
    rvCellX.shrink_to_fit();
    rvCellY.shrink_to_fit();

    // Leave
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        std::cout << "hipDeviceReset failed!" << std::endl;
        return 1;
    }

    return 0;
}
