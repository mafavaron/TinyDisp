
#include "hip/hip_runtime.h"
#include ""
#include "Config.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

int main(int argc, char** argv)
{

    // Get input parameters
    if (argc != 2) {
        std::cerr << "TinyDisp - A simple, local airflow visualizer" << std::endl;
        std::cerr << std::endl;
        std::cerr << "Usage:" << std::endl;
        std::cerr << std::endl;
        std::cerr << "    td <RunConfiguration>" << std::endl;
        std::cerr << std::endl;
        std::cerr << "Copyright 2020 by Servizi Territorio srl" << std::endl;
        std::cerr << "                  This is open-source software, covered by the MIT license" << std::endl;
        return 1;
    }
    std::string sCfgFile = argv[1];

    // Gather configuration (and meteo data)
    Config tCfg(sCfgFile);

    // Particle pool
    int iNumPart  = tCfg.GetParticlePoolSize();
    int iNextPart = 0;  // For indexing the generation circular buffer
    thrust::host_vector<int> ivPartTimeStamp(iNumPart); // Time stamp at emission time - for reporting - host-only
    thrust::device_vector<float> rvPartX(iNumPart);
    thrust::device_vector<float> rvPartY(iNumPart);
    thrust::device_vector<float> rvPartU(iNumPart);
    thrust::device_vector<float> rvPartV(iNumPart);

    // Initialize random number generator


    // Main loop: iterate over meteo data
    int iNumData = tCfg.GetNumMeteoData();
    for (auto i = 0; i < iNumData; i++) {

        // Get current meteorology
        int iTimeStamp;
        float rU;
        float rV;
        float rStdDevU;
        float rStdDevV;
        float rCovUV;
        bool lOk = tCfg.GetMeteo(i, iTimeStamp, rU, rV, rStdDevU, rStdDevV, rCovUV);

        // Emit new particles
        thrust::fill(ivPartTimeStamp.begin() + iNextPart, ivPartTimeStamp.begin() + iNextPart + tCfg.GetNumNewParticles(), iTimeStamp);
        thrust::fill(rvPartX.begin() + iNextPart, rvPartX.begin() + iNextPart + tCfg.GetNumNewParticles(), 0.0f);
        thrust::fill(rvPartY.begin() + iNextPart, rvPartY.begin() + iNextPart + tCfg.GetNumNewParticles(), 0.0f);
        thrust::fill(rvPartU.begin() + iNextPart, rvPartU.begin() + iNextPart + tCfg.GetNumNewParticles(), rU);
        thrust::fill(rvPartV.begin() + iNextPart, rvPartV.begin() + iNextPart + tCfg.GetNumNewParticles(), rV);
        iNextPart += tCfg.GetNumNewParticles();
        if (iNextPart >= iNumPart) {
            iNextPart = 0;
        }

        // Move particles

        // Count in cells

        // Inform users of the progress
        std::cout << iTimeStamp << ", " << rU << ", " << rV << ", " << rStdDevU << ", " << rStdDevV << ", " << rCovUV << std::endl;
    }

    // Leave
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        std::cout << "hipDeviceReset failed!" << std::endl;
        return 1;
    }

    return 0;
}

