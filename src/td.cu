
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <stdio.h>
#include <math.h>

int main(int argc, char** argv) {

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, NULL);

}
