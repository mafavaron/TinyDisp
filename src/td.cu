
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <stdio.h>
#include <math.h>

#define N (1<<20)

int main(int argc, char** argv) {

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, NULL);

	thrust::device_vector<float> dvec_x(N);
	thrust::device_vector<float> dvec_y(N);

	float *ptr_x = thrust::raw_pointer_cast(&dvec_x[0]);
	float *ptr_y = thrust::raw_pointer_cast(&dvec_y[0]);

	// Simulate particles emission
	hiprandGenerateUniform(gen, ptr_x, N);
	hiprandGenerateUniform(gen, ptr_y, N);

}
