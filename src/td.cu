
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <stdio.h>
#include <math.h>

#define N (1<<20)

int main(int argc, char** argv) {

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, NULL);

	thrust::device_vector<float> dvec_x(N);
	thrust::device_vector<float> dvec_y(N);

}
