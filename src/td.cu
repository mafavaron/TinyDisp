#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <stdio.h>
#include <math.h>
#include <string>
#include <iostream>
#include <fstream>

#include "cfg.h"
#include "meteodata.h"

#define N (1<<20)

int main(int argc, char** argv) {

	// Get input parameters
	if(argc != 2) {
		std::cerr << "td - The TinyDisp Particle Dispersion Model" << std::endl << std::endl;
		std::cerr << "Usage:" << std::endl << std::endl;
		std::cerr << "  [./]td <MeteoFileName>" << std::endl << std::endl;
		std::cerr << "Copyright 2019 by Mauri Favaron" << std::endl;
		std::cerr << "                  This is open-source software, covered by the MIT license" << std::endl << std::endl;
		return 1;
	}
	std::string sMetFileName = argv[1];

	// Read configuration
	std::ifstream fMeteoInputFile;
  fMeteoInputFile.open(sMetFileName, std::ios::in | std::ios::binary);
	Cfg tConfig = Cfg(fMeteoInputFile);
	if(tConfig.GetState() <= 0) {
		std::cerr << "Configuration file read failure" << std::endl;
		return 2;
	}
	int iRetCode = tConfig.Validate();
	if(iRetCode != 0) {
		std::cerr << "Configuration file validation failure, with code " << iRetCode << std::endl;
		return 3;
	}

	// Get emission data

	// Generate particle pool, and prepare for simulation

	// Main loop
	MeteoData met(tConfig.GetNumZ());
	while(true) {

		// Get meteo data
		iRetCode = met.Read(fMeteoInputFile, tConfig.GetNumZ());
		if(iRetCode != 0) break;
		std::cout << met.GetTimeStamp() << std::endl;

		// Move particles

		// Write particles to movie file, if requested

		// Count ground concentrations, if required, and write them to concentration file

	}

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 42ULL);

	thrust::device_vector<float> dvec_x(N);
	thrust::device_vector<float> dvec_y(N);

	float *ptr_x = thrust::raw_pointer_cast(&dvec_x[0]);
	float *ptr_y = thrust::raw_pointer_cast(&dvec_y[0]);

	// Simulate particles emission
	hiprandGenerateUniform(gen, ptr_x, N);
	hiprandGenerateUniform(gen, ptr_y, N);
	hiprandDestroyGenerator(gen);

	// Perform an aggregation function
	int insideCount = thrust::count_if(
		thrust::make_zip_iterator(thrust::make_tuple(dvec_x.begin(), dvec_y.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(dvec_x.end(), dvec_y.end())),
		[]__device__(const thrust::tuple<float, float> &el) {
			return(pow(thrust::get<0>(el), 2) + pow(thrust::get<1>(el), 2)) < 1.f;
		}
	);

	// Result...
	std::cout << "Pi = " << insideCount * 4.f / N << std::endl;

	// Leave
	return 0;

}
