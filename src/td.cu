#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <stdio.h>
#include <math.h>
#include <string>
#include <iostream>
#include <fstream>
#include <time.h>

#include "cfg.h"
#include "meteodata.h"

int main(int argc, char** argv) {

	// Get input parameters
	if(argc != 2) {
		std::cerr << "td - The TinyDisp Particle Dispersion Model" << std::endl << std::endl;
		std::cerr << "Usage:" << std::endl << std::endl;
		std::cerr << "  [./]td <MeteoFileName>" << std::endl << std::endl;
		std::cerr << "Copyright 2019 by Mauri Favaron" << std::endl;
		std::cerr << "                  This is open-source software, covered by the MIT license" << std::endl << std::endl;
		return 1;
	}
	std::string sMetFileName = argv[1];

	// Read configuration
	std::ifstream fMeteoInputFile;
  fMeteoInputFile.open(sMetFileName, std::ios::in | std::ios::binary);
	Cfg tConfig = Cfg(fMeteoInputFile);
	if(tConfig.GetState() <= 0) {
		std::cerr << "Configuration file read failure" << std::endl;
		return 2;
	}
	int iRetCode = tConfig.Validate();
	if(iRetCode != 0) {
		std::cerr << "Configuration file validation failure, with code " << iRetCode << std::endl;
		return 3;
	}

	// Get emission data
	// For the moment, assume a unit emission from a pointwise source places at domain center
	// and 5m height above ground.
	std::vector<double> rXs;
	rXs.push_back(tConfig.GetDomainCenterX());
	std::vector<double> rYs;
	rYs.push_back(tConfig.GetDomainCenterY());
	std::vector<double> rZs;
	rZs.push_back(5.0);
	std::vector<double> rEs;
	rEs.push_back(1.0);

	// Generate particle pool, and prepare for simulation
	int iPartIdx = -1;
	int iPartNum = 0;
	int N = tConfig.GetPartPoolSize();
	thrust::device_vector<float> rvdPartX(N);
	thrust::device_vector<float> rvdPartY(N);
	thrust::device_vector<float> rvdPartZ(N);
	thrust::device_vector<float> rvdPartU(N);
	thrust::device_vector<float> rvdPartV(N);
	thrust::device_vector<float> rvdPartW(N);
	thrust::device_vector<float> rvdPartQ(N);
	thrust::device_vector<float> rvdPartT(N);
	thrust::device_vector<float> rvdPartSh(N);
	thrust::device_vector<float> rvdPartSz(N);
	thrust::device_vector<float> rvdPartEmissionTime(N);	// -1.0 for not yet filled particles

	// Create random number generator, for use within loop
	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 42ULL);

	// Assign data vectors their initial values
	thrust::fill(rvdPartX.begin(), rvdPartX.end(), 0.f);
	thrust::fill(rvdPartY.begin(), rvdPartY.end(), 0.f);
	thrust::fill(rvdPartZ.begin(), rvdPartZ.end(), 0.f);
	thrust::fill(rvdPartU.begin(), rvdPartU.end(), 0.f);
	thrust::fill(rvdPartV.begin(), rvdPartV.end(), 0.f);
	thrust::fill(rvdPartW.begin(), rvdPartW.end(), 0.f);
	thrust::fill(rvdPartQ.begin(), rvdPartQ.end(), 0.f);
	thrust::fill(rvdPartT.begin(), rvdPartT.end(), 0.f);
	thrust::fill(rvdPartSh.begin(), rvdPartSh.end(), 0.f);
	thrust::fill(rvdPartSz.begin(), rvdPartSz.end(), 0.f);
	thrust::fill(rvdPartEmissionTime.begin(), rvdPartEmissionTime.end(), -1.f);

	// Main loop
	MeteoData met(tConfig.GetNumZ());
	while(true) {

		// Get meteo data
		iRetCode = met.Read(fMeteoInputFile, tConfig.GetNumZ());
		if(iRetCode != 0) break;

		// Print the current time stamp
		time_t iEpoch = (time_t)met.GetTimeStamp();
		struct tm * tStamp = gmtime(&iEpoch);
		char buffer[64];
		strftime(buffer, sizeof(buffer), "%Y-%m-%d %H:%M:%S", tStamp);
		std::cout << buffer << std::endl;

		// Generate new particles

		// Move particles

		// Write particles to movie file, if requested

		// Count ground concentrations, if required, and write them to concentration file

	}

	thrust::device_vector<float> dvec_x(N);
	thrust::device_vector<float> dvec_y(N);

	float *ptr_x = thrust::raw_pointer_cast(&dvec_x[0]);
	float *ptr_y = thrust::raw_pointer_cast(&dvec_y[0]);

	// Simulate particles emission
	hiprandGenerateUniform(gen, ptr_x, N);
	hiprandGenerateUniform(gen, ptr_y, N);
	hiprandDestroyGenerator(gen);

	// Perform an aggregation function
	int insideCount = thrust::count_if(
		thrust::make_zip_iterator(thrust::make_tuple(dvec_x.begin(), dvec_y.begin())),
		thrust::make_zip_iterator(thrust::make_tuple(dvec_x.end(), dvec_y.end())),
		[]__device__(const thrust::tuple<float, float> &el) {
			return(pow(thrust::get<0>(el), 2) + pow(thrust::get<1>(el), 2)) < 1.f;
		}
	);

	// Result...
	std::cout << "Pi = " << insideCount * 4.f / N << std::endl;

	// Leave
	return 0;

}
